#include "hip/hip_runtime.h"
/*
 *  dmv_gpu.cu -- Template for DMV GPU kernels
 *
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2013, Vasileios Karakasis
 */
#include <stdio.h>
#include "dmv.h"

/*
 *  Utility function to get the thread ID within the
 *  global working space.
 */
__device__ int get_global_tid()
{
    return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y +
        blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Utility function to get the thread ID within the
 *  local/block working space.
 */
__device__ int get_local_tid()
{
    return blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Naive kernel
 */
__global__ void dmv_gpu_naive(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    int i, j, a_j, limitj;
    value_t _yi = 0;

    j = blockIdx.x*blockDim.x;
    limitj = j + blockDim.x;
    i = threadIdx.x + blockIdx.y*blockDim.x;
    a_j = i*n + j;

    for (; j < limitj; ++j, ++a_j)
        _yi += a[a_j]*x[j];
    atomicAdd(&y[i], _yi);
}
/*
 *  Naive kernel 1-dim grid
 */
/*
__global__ void dmv_gpu_naive(const value_t *a, const value_t *x, value_t *y,
                              size_t n
{
   int tid = get_global_tid();
   size_t j, a_j;
   register value_t _yi = 0;
   if (tid < n) {
       for (j = 0, a_j=tid*n; j < n; ++j, ++a_j) {
           _yi += a[a_j]*x[j];
       }
       y[tid] = _yi;
   }
}
 */

/*
 *  Coalesced memory acceses
 */
__global__ void dmv_gpu_coalesced(const value_t *a, const value_t *x,
                                  value_t *y, size_t n)
{
    int i, j, a_j, limitj;
    value_t _yi = 0;

    j = blockIdx.x*blockDim.x;
    limitj = j + blockDim.x;
    i = threadIdx.x + blockIdx.y*blockDim.x;
    a_j = j*n + i;

    for (; j < limitj; ++j, a_j += n)
        _yi += a[a_j]*x[j];
    atomicAdd(&y[i], _yi);
}
/*
 *  Coalesced memory acceses 1-dim grid
 */
/*
__global__ void dmv_gpu_coalesced(const value_t *a, const value_t *x,
                                  value_t *y, size_t n)
{
   int tid = get_global_tid();
   size_t j, a_j;
   register value_t _yi = 0;
   if (tid < n) {
       for (j = 0, a_j=tid; j < n; ++j, a_j += n) {
           _yi += a[a_j]*x[j];
       }
       y[tid] = _yi;
   }
}
 */


/*
 *  Use of shared memory
 */
__global__ void dmv_gpu_shmem(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    int i, j, a_j, limitj;
    value_t _yi = 0;
    extern __shared__ value_t sh_x[];

    j = blockIdx.x*blockDim.x;
    limitj = j + blockDim.x;
    i = threadIdx.x + blockIdx.y*blockDim.x;
    a_j = j*n + i;

    sh_x[threadIdx.x] = x[j + threadIdx.x];
    __syncthreads();
    for (; j < limitj; ++j, a_j += n)
        _yi += a[a_j]*x[j];
    atomicAdd(&y[i], _yi);
}
/*
 *  Use of shared memory 1-dim grid
 */
/*
__global__ void dmv_gpu_shmem(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    int loc_tid = get_local_tid();
    int tid = get_global_tid();
    size_t i, j, a_j;
    register value_t _yi = 0;
    extern __shared__ value_t sh_x[];
    for (i = 0; i < n; i += blockDim.x) {
        sh_x[loc_tid] = x[i+loc_tid];
        __syncthreads();
        for (j = 0, a_j = tid + i*n; j < blockDim.x; ++j, a_j += n) {
            _yi += a[a_j]*sh_x[j];
        }
        __syncthreads();
    }
    y[tid] = _yi;
}
 */

